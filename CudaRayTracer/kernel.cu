#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <chrono>

#include "utility.h"

#include "hittable.h"
#include "hittable_list.h"
#include "sphere.h"
#include "camera.h"
#include "material.h"
#include "renderer.h"

#include "model_loader.h"

#include "scene.h"

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include "hip/hip_runtime.h"
#include ""

class Scene;


__global__ void free_world(Hittable** d_world, Camera** d_camera) {
    delete* d_world;
    delete* d_camera;
}

__global__ void init_camera(Camera** d_camera, uint32_t width, float aspect_ratio) {
    if (blockIdx.x == 0 && threadIdx.x == 0) {
		float vfov = 25.0f;
		vec3 look_from = vec3(8.f, 5.f, -6.0f);
		vec3 look_at = vec3(-4, 0, 4);

		float defocus_angle = 0.6f;
		float focus_distance = glm::length(look_from - vec3(0,0,0));

		*d_camera = new Camera(width, look_from, look_at, vec3(0, 1, 0),
            vfov, aspect_ratio, defocus_angle, focus_distance);
    }
}

int main()
{
    const float ASPECT_RATIO = 16.0f / 9.0f;
    uint32_t IMAGE_WIDTH = 1000;
    uint32_t IMAGE_HEIGHT = 1000;
    const uint32_t spp = 10;
    const uint32_t tx = 8;
    const uint32_t ty = 8;

    std::cerr << "Rendering a " << IMAGE_WIDTH << "x" << IMAGE_HEIGHT << " image with " << spp << " samples per pixel ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    // Initialize frame buffer
    uint32_t num_pixels = IMAGE_WIDTH * IMAGE_HEIGHT;
    size_t framebuffer_size = num_pixels * sizeof(vec3);
    vec3* framebuffer;
    checkCudaErrors(hipMallocManaged((void**)&framebuffer, framebuffer_size));

    // Initialize camera on GPU
    Camera** d_camera;
    checkCudaErrors(hipMalloc((void**)&d_camera, sizeof(Camera*)));
    init_camera<<<1, 1>>>(d_camera, IMAGE_WIDTH, ASPECT_RATIO);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // Create list of Spheres on GPU
    Hittable** d_world;
    checkCudaErrors(hipMalloc((void**)&d_world, sizeof(Hittable*)));
    Scene scene;
    //scene.launch_random_scene_kernel(d_world);
    scene.launch_triangles_scene(d_world);
    std::clog << "- Scene Created\n";

    // Create renderer
    Renderer renderer(IMAGE_WIDTH, IMAGE_HEIGHT, tx, ty, spp);
 
    // Start rendering
    clock_t start, stop;
    start = clock();
    
    renderer.launch_kernel_render(framebuffer, d_camera, d_world);

    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "\nRender time: " << timer_seconds << " seconds.\n";

    // Export to png
    export_framebuffer_to_png(framebuffer, IMAGE_WIDTH, IMAGE_HEIGHT, "output.png");

    free_world<<<1, 1>>>(d_world, d_camera);
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_camera));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(framebuffer));

    hipDeviceReset();

    return 0;
}
